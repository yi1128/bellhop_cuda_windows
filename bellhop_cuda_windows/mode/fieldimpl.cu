#include "hip/hip_runtime.h"
// For CUDA

#include "hip/hip_runtime.h"
#include ""

#include "../trace.hpp"
#include "fieldimpl.h"

#ifdef BHC_BUILD_CUDA
namespace bhc { namespace mode {

#define NUM_THREADS 256
#define LAUNCH_BOUNDS __launch_bounds__(NUM_THREADS, 1)

    using GENCFG = CfgSel<BHCGENRUN, BHCGENINFL, BHCGENSSP>;
    
    template<typename CFG, bool O3D, bool R3D> __global__ void
        FieldModesKernel(bhcParams<BHCGENO3D> params, bhcOutputs<BHCGENO3D, BHCGENR3D> outputs,
            ErrState* errState);

    template<typename CFG, bool O3D, bool R3D> __global__ void
        FieldModesKernel(
            bhcParams<BHCGENO3D> params,
            bhcOutputs<BHCGENO3D, BHCGENR3D> outputs,
            ErrState* errState)
    {
        for (int32_t job = blockIdx.x * blockDim.x + threadIdx.x; true;
            job += gridDim.x * blockDim.x) {
            RayInitInfo rinit;
            if (!GetJobIndices<BHCGENO3D>(rinit, job, params.Pos, params.Angles)) break;

            MainFieldModes<GENCFG, BHCGENO3D, BHCGENR3D>(
                rinit, outputs.uAllSources, params.Bdry, params.bdinfo, params.refl,
                params.ssp, params.Pos, params.Angles, params.freqinfo, params.Beam,
                params.sbp, outputs.eigen, outputs.arrinfo, errState);
        }
    }



    template<typename CFG, bool O3D, bool R3D> void RunFieldModesImpl(
        bhcParams<BHCGENO3D>& params, bhcOutputs<BHCGENO3D, BHCGENR3D>& outputs)
    {
        ErrState* errState;
        checkCudaErrors(hipMallocManaged(&errState, sizeof(ErrState)));
        ResetErrState(errState);
        FieldModesKernel<GENCFG, BHCGENO3D, BHCGENR3D> << <GetInternal(params)->d_multiprocs, NUM_THREADS >> > (params, outputs, errState);

        /*
        std::string __str = std::string("FieldModesKernel<") + std::string(BHCGENRUN) + std::string(",") +
            std::string(BHCGENINFL) + std::string(",") + std::string(BHCGENSSP) + std::string(",") +
            std::string(O3D) + std::string(",") + std::string(BHCGENR3D) + std::string(">");
        char* __cstr = __str.data();
        */
        syncAndCheckKernelErrors("FieldModesKernel<BHCGENRUN, BHCGENINFL, BHCGENSSP, "
            "BHCGENO3D, BHCGENR3D>");
        
        CheckReportErrors(GetInternal(params), errState);
        checkCudaErrors(hipFree(errState));
    }
    
    
    void RunFieldModesImplLaunch(
        bhcParams<BHCGENO3D>& params, bhcOutputs<BHCGENO3D, BHCGENR3D>& outputs)
    {
        RunFieldModesImpl<GENCFG, BHCGENO3D, BHCGENR3D>(params, outputs);
    }
    
/*
using GENCFG = CfgSel<BHCGENRUN, BHCGENINFL, BHCGENSSP>;


template<typename CFG, bool O3D, bool R3D> __global__ void 
FieldModesKernel(bhcParams<O3D> params, bhcOutputs<O3D, R3D> outputs,
    ErrState *errState);

template<> __global__ void 
FieldModesKernel<GENCFG, BHCGENO3D, BHCGENR3D>(
    bhcParams<BHCGENO3D> params,
    bhcOutputs<BHCGENO3D, BHCGENR3D> outputs,
    ErrState *errState)
{
    for(int32_t job = blockIdx.x * blockDim.x + threadIdx.x; true;
        job += gridDim.x * blockDim.x) {
        RayInitInfo rinit;
        if(!GetJobIndices<BHCGENO3D>(rinit, job, params.Pos, params.Angles)) break;

        MainFieldModes<GENCFG, BHCGENO3D, BHCGENR3D>(
            rinit, outputs.uAllSources, params.Bdry, params.bdinfo, params.refl,
            params.ssp, params.Pos, params.Angles, params.freqinfo, params.Beam,
            params.sbp, outputs.eigen, outputs.arrinfo, errState);
    }
}

template<> void RunFieldModesImpl<GENCFG, BHCGENO3D, BHCGENR3D>(
    bhcParams<BHCGENO3D> &params,
    bhcOutputs<BHCGENO3D, BHCGENR3D> &outputs)
{
    ErrState *errState;
    checkCudaErrors(hipMallocManaged(&errState, sizeof(ErrState)));
    ResetErrState(errState);
    FieldModesKernel<GENCFG, BHCGENO3D, BHCGENR3D>
        <<<GetInternal(params)->d_multiprocs, NUM_THREADS>>>(params, outputs, errState);
    syncAndCheckKernelErrors("FieldModesKernel<@BHCGENRUN@, @BHCGENINFL@, @BHCGENSSP@, "
                             "@BHCGENO3D@, @BHCGENR3D@>");
    CheckReportErrors(GetInternal(params), errState);
    checkCudaErrors(hipFree(errState));
}
*/
}} // namespace bhc::mode
#endif